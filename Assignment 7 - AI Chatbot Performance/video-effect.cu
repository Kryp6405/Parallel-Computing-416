
#include <hip/hip_runtime.h>
#include <vector>   // vector

/*  use this to set the block size of the kernel launches.
    CUDA kernels will be launched with block size blockDimSize by blockDimSize. */
constexpr int blockDimSize = 32;

/*  your job is to write convolveGPU:
    convolveGPU will be called with blockSize blockDimSize x blockDimSize 
    and gridsize ⌈height/blockDimSize⌉x⌈width/blockDimSize⌉.
    Each thread may have to compute more than one pixel. You will need to stride the computation.
    Look at convolveCPU below for more info.
*/
__global__ void convolveGPU(float const* in, float *out, int width, int height, float const* kernel, int kernelWidth, int kernelHeight) {
    // Calculate the thread's global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Stride values to cover the entire image
    int strideX = gridDim.x * blockDim.x;
    int strideY = gridDim.y * blockDim.y;

    // Calculate half sizes of the kernel
    int halfKernelHeight = kernelHeight / 2;
    int halfKernelWidth = kernelWidth / 2;

    // Channel indices
    const int redChannel = 2;
    const int greenChannel = 1;
    const int blueChannel = 0;

    // Loop over the image pixels with striding
    for (int i = y; i < height; i += strideY) {
        for (int j = x; j < width; j += strideX) {
            // Ignore border pixels where the kernel would go out of bounds
            if (i >= halfKernelHeight && i < (height - halfKernelHeight) && j >= halfKernelWidth && j < (width - halfKernelWidth)) {
                // Initialize accumulators for each color channel
                float redDot = 0.0f;
                float greenDot = 0.0f;
                float blueDot = 0.0f;

                // Apply the kernel to the current pixel
                for (int m = -halfKernelHeight; m <= halfKernelHeight; m++) {
                    for (int n = -halfKernelWidth; n <= halfKernelWidth; n++) {
                        int neighborRow = i + m;
                        int neighborCol = j + n;

                        // Calculate indices for the input image and kernel
                        int imageIdx = (neighborRow * width + neighborCol) * 3;
                        int kernelIdx = (m + halfKernelHeight) * kernelWidth + (n + halfKernelWidth);

                        float kernelValue = kernel[kernelIdx];

                        // Accumulate the weighted sum for each color channel
                        blueDot  += in[imageIdx + blueChannel]  * kernelValue;
                        greenDot += in[imageIdx + greenChannel] * kernelValue;
                        redDot   += in[imageIdx + redChannel]   * kernelValue;
                    }
                }

                // Write the result to the output image
                int outputIdx = (i * width + j) * 3;
                out[outputIdx + blueChannel]  = blueDot;
                out[outputIdx + greenChannel] = greenDot;
                out[outputIdx + redChannel]   = redDot;
            }
        }
    }
}

/* A CPU example of the convolve kernel */
void convolveCPU(float const* in, float *out, int width, int height, float const* kernel, int kernelWidth, int kernelHeight) {
    const int halfKernelHeight = kernelHeight/2;
    const int halfKernelWidth = kernelWidth/2;
    const int redChannel = 2;
    const int greenChannel = 1;
    const int blueChannel = 0;

    /* point-wise loop over the image pixels */
    for (int i = halfKernelHeight; i < height-halfKernelHeight; i += 1) {
        for (int j = halfKernelWidth; j < width-halfKernelWidth; j += 1) {

            /* compute dot product of kernel and sub-image */
            float redDot = 0.0f, greenDot = 0.0f, blueDot = 0.0f;
            for (int k = -halfKernelHeight; k <= halfKernelHeight; k += 1) {
                for (int l = -halfKernelWidth; l <= halfKernelWidth; l += 1) {

                    /* add in[i+k][j+l]*kernel[k][l] to dot product for red, green, and blue */
                    redDot += in[(i+k)*width*3 + (j+l)*3 + redChannel] * kernel[(k+halfKernelHeight)*kernelWidth + (l+halfKernelWidth)];
                    greenDot += in[(i+k)*width*3 + (j+l)*3 + greenChannel] * kernel[(k+halfKernelHeight)*kernelWidth + (l+halfKernelWidth)];
                    blueDot += in[(i+k)*width*3 + (j+l)*3 + blueChannel] * kernel[(k+halfKernelHeight)*kernelWidth + (l+halfKernelWidth)];
                
                }
            }

            /* set out[i][j] to dot product */
            out[i*width*3 + j*3 + redChannel] = redDot;
            out[i*width*3 + j*3 + greenChannel] = greenDot;
            out[i*width*3 + j*3 + blueChannel] = blueDot;

        }
    }
}

/* call the convolveGPU function on each frame */
float convolveFrames(std::vector<float *> const& framesIn, std::vector<float *> &framesOut, int width, int height, float const* kernel, int kernelWidth, int kernelHeight,
    hipStream_t *streams, int numStreams, int gridSizeX, int gridSizeY) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blockSize (blockDimSize, blockDimSize);
    dim3 gridSize (gridSizeX, gridSizeY);

    hipEventRecord(start, 0);
    for (int i = 0; i < framesIn.size(); i += 1) {
        convolveGPU<<<gridSize, blockSize, 0, streams[i % numStreams]>>>(framesIn.at(i), framesOut.at(i), width, height, kernel, kernelWidth, kernelHeight);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return (elapsed / 1000.0f);
}
